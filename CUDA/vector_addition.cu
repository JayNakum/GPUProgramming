#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void d_vectorAdd(float* vecA, float* vecB, float* resultVec, int n)
{
    // CUDA kernel definition
    int i = threadIdx.x + (blockIdx.x * blockDim.x);
    if (i < n)
        resultVec[i] = vecA[i] + vecB[i];
}

// host program
void h_vectorAdd(float* h_vecA, float* h_vecB, float* h_resultVec, int n)
{
    int size = n * sizeof(float);
    float *d_vecA = NULL, *d_vecB = NULL, *d_resultVec = NULL;
    hipError_t err = hipSuccess;

    printf("Allocating Memory.\n");

    err = hipMalloc((void **)&d_vecA, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "(%s) Failed to allocate device vector d_vecA.\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMalloc((void **)&d_vecB, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "(%s) Failed to allocate device vector d_vecB.\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    err = hipMalloc((void **)&d_resultVec, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "(%s) Failed to allocate device vector d_resultVec.\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Copying input data from host to device.\n");
    
    err = hipMemcpy(d_vecA, h_vecA, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "(%s) Failed to copy vecA.\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_vecA, h_vecB, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "(%s) Failed to copy vecB.\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock, -1) / threadsPerBlock;

    printf("CUDA kernel launch with %d blocks of %d threads.\n", blocksPerGrid, threadsPerBlock);
    
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_vecA, d_vecB, d_resultVec, n);

    err = hipGetLastError();
    if (err != hipSuccess)
    {
        fprintf(stderr, "(%s) Failed to launch vectorAdd() kernel.\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Copying output data from device to host.\n");

    err = hipMemcpy(h_resultVec, d_resultVec, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "(%s) Failed to copy resultVec.\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    hipFree(d_vecA);
    hipFree(d_vecB);
    hipFree(d_resultVec);

    // Result Verification
    for(int i = 0 ; i < n ; ++i)
    {
        if(fabs(h_vecA[i] + h_vecB[i] - h_resultVec[i]) > 1e-5)
        {
            printf("Test FAILED\n");
            fprintf(stderr, "Result verification failed at element %d.\n", i);
            exit(EXIT_FAILURE);
        }
    }
    printf("Test PASSED\n");
}
